#include "hip/hip_runtime.h"
#include "ccomplex.cuh"

#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

#include "ckernels.h"

// DEVICE FUNCTIONS

__device__ __host__ Complex cAdd(Complex a, Complex b) {
	/*
	Add the real and imaginary components of two complex numbers.
	*/
	Complex c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}

__device__ __host__ Complex cConvolveKernelReal(int i, Complex* a, long dim1, double* kernel, long kernel_size) {
	/*
	Convolve the real component of array [a] with kernel [kernel].
	*/
	long kernel_half_size = (kernel_size - 1) / 2;
	Complex new_value;
	new_value.x = 0;
	new_value.y = 0;
	for (int kj = 0; kj < kernel_size; kj++) {
		for (int ki = 0; ki < kernel_size; ki++) {
			new_value.x += a[i + ((ki - kernel_half_size) + (kj - kernel_half_size)*dim1)].x * kernel[(kj*kernel_size) + ki];
		}
	}
	return new_value;
}

__device__ __host__ double cGetAmplitude(Complex a) {
	/*
	Get the amplitude of the complex number [a].
	*/
	double abs = sqrt(pow(a.x, 2) + pow(a.y, 2));
	return abs;
}

__device__ __host__ double cGetPhase(Complex a) {
	/*
	Get the phase of the complex number [a].
	*/
	double phase = atan2(a.y, a.x);
	return phase;
}

__device__ __host__ long cGet1DIndexFrom2DXY(long2 xy, long dim1) {
	/*
	Given a pair of coordinates [xy] and array x dimension [dim1], find the corresponding 1D index.
	*/
	long index = (xy.y*dim1) + xy.x;
	return index;
}

__device__ __host__ long2 cGet2DXYFrom1DIndex(long index, long dim1) {
	/*
	Given a 1D index [index] and array x dimension [dim1], find the corresponding pair of coordinates xy.
	*/
	long2 xy;
	xy.x = index % dim1;
	xy.y = (long)(index / dim1);
	return xy;
}

__device__ __host__ quadrant cGet2DQuadrantFrom1DIndex(long index, long dim1, long x_split, long y_split) {
	/*
	Given a 1D index [index] and array x dimension [dim1], find the quadrant in which the index would lie 
	in a 2D array given the x/y split positions [x_split, y_split].
	*/
	long2 xy = cGet2DXYFrom1DIndex(index, dim1);
	if (xy.x < x_split) {
		if (xy.y < y_split) {
			return Q1;
		}
		else {
			return Q3;
		}
	}
	else {
		if (xy.y < y_split) {
			return Q2;
		}
		else {
			return Q4;
		}
	}
}

__device__ __host__ Complex cScale(Complex a, double s) {
	Complex c;
	c.x = a.x * s;
	c.y = a.y * s;
	return c;
}

__device__ __host__ Complex cSub(Complex a, Complex b) {
	/*
	Subtract a complex number [a] from a complex number [b].
	*/
	Complex c;
	c.x = a.x - b.x;
	c.y = a.y - b.y;
	return c;
}

// GLOBAL FUNCTIONS

__global__ void cAdd2D(Complex* a, Complex* b, long size) {
	/*
	Add the numbers from complex array [a] with [size] elements to complex array [b] pointwise.
	*/
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	// this is required as one thread may need to do multiple 
	// computations, i.e. if numThreads < size
	for (int i = threadID; i < size; i += numThreads) {
		a[i] = cAdd(a[i], b[i]);
	}
}

__global__ void cConvolveKernelReal2D(Complex* a, Complex* b, long dim1, double* kernel, long kernel_size) {
	/*
	Convolve kernel [kernel] of dimension [kernel_size] with complex array [a] of x dimension [dim1] and store 
	the result in [b].
	*/
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	long size = dim1*dim1;
	// this is required as one thread may need to do multiple 
	// computations, i.e. if numThreads < size
	for (int i = threadID; i < size; i += numThreads) {
		long2 xy = cGet2DXYFrom1DIndex(i, dim1);
		long kernel_half_size = (kernel_size - 1) / 2;
		if (xy.x >= kernel_half_size && xy.x < dim1 - kernel_half_size && xy.y >= kernel_half_size && xy.y < dim1 - kernel_half_size) {
			b[i] = cConvolveKernelReal(i, a, dim1, kernel, kernel_size);
		}
	}
}

__global__ void cFftShift2D(Complex* a, Complex* b, long dim) {
	/*
	Perform an fftshift on a complex array [a] to yield [b]. This routine only handles arrays with dimensions of equal size [dim].
	*/
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	long size = dim*dim;
	long Q1_offset, Q2_offset, Q3_offset, Q4_offset;
	if (dim % 2 == 0) {
		Q1_offset = (dim*((dim) / 2.)) + ((dim) / 2);
		Q2_offset = (dim*((dim) / 2)) - ((dim) / 2);
		Q3_offset = -(dim*((dim) / 2)) + ((dim) / 2);
		Q4_offset = -(dim*((dim) / 2)) - ((dim) / 2);
	}
	else {
		Q1_offset = (dim*ceil(dim / 2.)) + ceil(dim / 2.);
		Q2_offset = (dim*ceil(dim / 2.)) - floor(dim / 2.);
		Q3_offset = -(dim*floor(dim / 2.)) + ceil(dim / 2.);
		Q4_offset = -(dim*floor(dim / 2.)) - floor(dim / 2.);
	}

	long x_split = floor(dim / 2.);
	long y_split = floor(dim / 2.);

	for (int i = threadID; i < size; i += numThreads) {
		switch (cGet2DQuadrantFrom1DIndex(i, dim, x_split, y_split)) {
		case Q1:
			b[i] = a[i + Q1_offset];
			break;
		case Q2:
			b[i] = a[i + Q2_offset];
			break;
		case Q3:
			b[i] = a[i + Q3_offset];
			break;
		case Q4:
			b[i] = a[i + Q4_offset];
			break;
		}
	}
}

__global__ void cIFftShift2D(Complex* a, Complex* b, long dim) {
	/*
	Perform an ifftshift on a complex array [a] to yield [b]. This routine only handles arrays with dimensions of equal size [dim].
	*/
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	long size = dim*dim;
	long Q1_offset, Q2_offset, Q3_offset, Q4_offset;
	if (dim % 2 == 0) {
		Q1_offset = (dim*((dim) / 2)) + ((dim) / 2);
		Q2_offset = (dim*((dim) / 2)) - ((dim) / 2);
		Q3_offset = -(dim*((dim) / 2)) + ((dim) / 2);
		Q4_offset = -(dim*((dim) / 2)) - ((dim) / 2);
	}
	else {
		Q1_offset = (dim*floor(dim / 2.)) + floor(dim / 2.);
		Q2_offset = (dim*floor(dim / 2.)) - ceil(dim / 2.);
		Q3_offset = -(dim*ceil(dim / 2.)) + floor(dim / 2.);
		Q4_offset = -(dim*ceil(dim / 2.)) - ceil(dim / 2.);
	}
	long x_split = ceil(dim / 2.);
	long y_split = ceil(dim / 2.);

	for (int i = threadID; i < size; i += numThreads) {
		switch (cGet2DQuadrantFrom1DIndex(i, dim, x_split, y_split)) {
		case Q1:
			b[i] = a[i + Q1_offset];
			break;
		case Q2:
			b[i] = a[i + Q2_offset];
			break;
		case Q3:
			b[i] = a[i + Q3_offset];
			break;
		case Q4:
			b[i] = a[i + Q4_offset];
			break;
		}
	}
}

__global__ void cScale2D(Complex *a, double scale, long size) {
	/*
	Scale complex array [a] with [size] elements by [scale] pointwise.
	*/
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	// this is required as one thread may need to do multiple 
	// computations, i.e. if numThreads < size
	for (int i = threadID; i < size; i += numThreads) {
		a[i] = cScale(a[i], scale);
	}
}

__global__ void cSub2D(Complex* a, Complex* b, long size) {
	/* 
	Subtract array [b] with [size] elements from array [a] pointwise.
	*/
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	// this is required as one thread may need to do multiple 
	// computations, i.e. if numThreads < size
	for (int i = threadID; i < size; i += numThreads) {
		a[i] = cSub(a[i], b[i]);
	}
}

__global__ void cSetComplexRealAsAmplitude(Complex *a, long size) {
	/*
	Set the real component of array [a] with [size] elements to the amplitude and zero the  
	imaginary part.
	*/
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	// this is required as one thread may need to do multiple 
	// computations, i.e. if numThreads < size
	for (int i = threadID; i < size; i += numThreads) {
		a[i].x = cGetAmplitude(a[i]);
		a[i].y = 0;
	}
}
