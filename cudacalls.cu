#include "hip/hip_runtime.h"
#include "cudacalls.cuh"
#include "ccube.h"

hipError_t cudaCompareArray2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, int** in, int* out, long index, long n_slices, long n_spaxels_per_slice) {
	cCompareArray2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, index, n_slices, n_spaxels_per_slice);
	return hipGetLastError();
}

hipError_t cudaGetSpaxelData2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex** in, Complex** out, long n_slices, long n_spaxels_per_slice) {
	cGetSpaxelData2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, n_slices, n_spaxels_per_slice);
	return hipGetLastError();
}

hipError_t cudaFftShift2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in, Complex* out, long x_size) {
	cFftShift2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, x_size);
	return hipGetLastError();
}

hipError_t cudaIFftShift2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in, Complex* out, long x_size) {
	cIFftShift2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, x_size);
	return hipGetLastError();
}

hipError_t cudaMakeBitmask2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex** in, int** out, long n_slices, long n_spaxels_per_slice) {
	cMakeBitmask2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, n_slices, n_spaxels_per_slice);
	return hipGetLastError();
}

hipError_t cudaScale2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* data, double constant, long memsize) {
	cScale2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(data, constant, memsize);
	return hipGetLastError();
}

hipError_t cudaSetComplexRealAsAmplitude2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* a, long size) {
	cSetComplexRealAsAmplitude2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(a, size);
	return hipGetLastError();
}

hipError_t cudaPolySub2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex** in, int** mask, Complex** coeffs, long n_coeffs, int* wavelengths, long n_slices, long n_spaxels_per_slice) {
	cPolySub2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, mask, coeffs, n_coeffs, wavelengths, n_slices, n_spaxels_per_slice);
	return hipGetLastError();
}

hipError_t cudaTranslate2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* a, double2 translation, long x_size) {
	cTranslate2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(a, translation, x_size);
	return hipGetLastError();
}
