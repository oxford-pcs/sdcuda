#include "hip/hip_runtime.h"
#include "cudacalls.cuh"
#include "ccube.h"

hipError_t cudaCompareArray2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, int** in, int* out, long index, long n_slices, long n_spaxels_per_slice) {
	cCompareArray2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, index, n_slices, n_spaxels_per_slice);
	return hipGetLastError();
}

hipError_t cudaDivideByRealComponent2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in1, Complex* in2, Complex* out, long n_spaxels_per_slice) {
	cDivideByRealComponent2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in1, in2, out, n_spaxels_per_slice);
	return hipGetLastError();
}

hipError_t cudaGetSpaxelData2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex** in, Complex** out, long n_slices, long n_spaxels_per_slice) {
	cGetSpaxelData2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, n_slices, n_spaxels_per_slice);
	return hipGetLastError();
}

hipError_t cudaFftShift2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in, Complex* out, long x_size) {
	cFftShift2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, x_size);
	return hipGetLastError();
}

hipError_t cudaIFftShift2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in, Complex* out, long x_size) {
	cIFftShift2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, x_size);
	return hipGetLastError();
}

hipError_t cudaMakeBitmask2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex** in, int** out, long n_slices, long n_spaxels_per_slice) {
	cMakeBitmask2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, n_slices, n_spaxels_per_slice);
	return hipGetLastError();
}

hipError_t cudaMultiplyHadamard2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in1, Complex* in2, Complex* out, long n_spaxels_per_slice) {
	cMultiplyHadamard2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in1, in2, out, n_spaxels_per_slice);
	return hipGetLastError();
}

hipError_t cudaScale2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in, double factor, long memsize) {
	cScale2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, factor, memsize);
	return hipGetLastError();
}

hipError_t cudaSetComplexRealAsAmplitude2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in, long size) {
	cSetComplexRealAsAmplitude2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, size);
	return hipGetLastError();
}

hipError_t cudaPolySub2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex** in, int** mask, Complex** coeffs, long n_coeffs, int* wavelengths, long n_slices, long n_spaxels_per_slice) {
	cPolySub2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, mask, coeffs, n_coeffs, wavelengths, n_slices, n_spaxels_per_slice);
	return hipGetLastError();
}

hipError_t cudaTranslate2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in, double2 translation, long x_size) {
	cTranslate2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, translation, x_size);
	return hipGetLastError();
}
