#include "hip/hip_runtime.h"
#include "cudacalls.cuh"
#include "ccube.h"

hipError_t cudaSubtractPoly(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex** in, Complex* coeffs, long n_coeffs, int* wavelengths, long n_slices, long n_spaxels) {
	cSubtractPoly << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, coeffs, n_coeffs, wavelengths, n_slices, n_spaxels);
	return hipGetLastError();
}

hipError_t cudaGetSpaxelData2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex** in, Complex* out, long n_slices, long n_spaxels) {
	cGetSpaxelData2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, n_slices, n_spaxels);
	return hipGetLastError();
}

hipError_t cudaFftShift2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in, Complex* out, long x_size) {
	cFftShift2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, x_size);
	return hipGetLastError();
}

hipError_t cudaIFftShift2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* in, Complex* out, long x_size) {
	cIFftShift2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(in, out, x_size);
	return hipGetLastError();
}

hipError_t cudaScale2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* data, double constant, long memsize) {
	cScale2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(data, constant, memsize);
	return hipGetLastError();
}

hipError_t cudaSetComplexRealAsAmplitude(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* a, long size) {
	cSetComplexRealAsAmplitude << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(a, size);
	return hipGetLastError();
}

hipError_t cudaTranslate2D(int nCUDABLOCKS, int nCUDATHREADSPERBLOCK, Complex* a, double2 translation, long x_size) {
	cTranslate2D << <nCUDABLOCKS, nCUDATHREADSPERBLOCK >> >(a, translation, x_size);
	return hipGetLastError();
}
